#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Matrix multiplication: C = A * B.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
  
  int tile_width = 32;
  __shared__ float Mds[32][32];
  __shared__ float Nds[32][32];

  // Identify the row and column of Pd element to compute
  int Row = blockIdx.y * tile_width + threadIdx.y;
  int Col = blockIdx.x * tile_width + threadIdx.x;
  int Width = P.width;

  float pValue = 0;
  // Loop over Md and Nd tiles to compute Pd element.
   
  for (int m = 0; m < Width/tile_width; m++) {
    Mds[threadIdx.y][threadIdx.x] = M.elements[Row*Width + (m * tile_width + threadIdx.x)];
    Nds[threadIdx.y][threadIdx.x] = N.elements[Col + (m * tile_width + threadIdx.y) * Width];

    // Synchronize
    __syncthreads();
    for (int k = 0; k < tile_width; k++) {
      pValue += Mds[threadIdx.y][k] * Nds[k][threadIdx.y];
    __syncthreads();
    }
  }
  P.elements[Row*Width+Col] = pValue;
  
  /* 
  for (int k = 0; k < Width; ++k) {
    pValue += M.elements[Row*Width+k] * N.elements[k*Width+Col];
  }
  P.elements[Row*Width + Col] = pValue; 
  */
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
